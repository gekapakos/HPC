#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"

PGM_IMG contrast_enhancement_g(PGM_IMG img_in)  {
    PGM_IMG result;
    int hist[256];

    result.w = img_in.w;
    result.h = img_in.h;
    result.img = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));

    histogram(hist, img_in.img, img_in.h * img_in.w, 256);
    histogram_equalization(result.img,img_in.img,hist,result.w*result.h, 256);

    /*for(int i = 0;i < 256; i++) {
        printf("%d ", hist[i]);
    }*/

    return result;
}

PGM_IMG contrast_enhancement_GPU(PGM_IMG img_in)  {
    hipEvent_t startCuda, stopCuda, startCuda1, stopCuda1;
    PGM_IMG gpuResult;
    PGM_IMG result;
    float millisecondsTransfers = 0, time, millisecondsTransfers1 = 0;
    //int hist[256];
    int t_hist[256], t_hist0[128], t_hist1[128];
    int *d_hist, *A0_hist, *A1_hist, *A2_hist, *A3_hist, *A4_hist, *A5_hist, *A6_hist, *A7_hist;
    // unsigned char * d_ImgIn;
    unsigned char * img_A0,* img_A1,* img_A2,* img_A3,* img_A4,* img_A5,* img_A6,* img_A7, *img_A[2];
    int SegSize = (img_in.w * img_in.h)/2;
    int t0_hist[256], t1_hist[256];
    hipStream_t stream[2];
    

    result.w = img_in.w;
    result.h = img_in.h;
    result.img = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));

    hipMallocManaged(&gpuResult.img, result.w * result.h * sizeof(unsigned char));
    gpuResult.w = img_in.w;
    gpuResult.h = img_in.h;

    hipEventCreate(&startCuda);
    hipEventCreate(&stopCuda);
    hipEventCreate(&startCuda1);
    hipEventCreate(&stopCuda1);

    hipEventRecord(startCuda);


    /*hipError_t err = hipMalloc((void **)&gpuResult.img, gpuResult.w * gpuResult.h * sizeof(unsigned char));
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
        if (gpuResult.img) hipFree(gpuResult.img);
        return(gpuResult);
    }*/
    // hipError_t err = hipMalloc((void **)&d_ImgIn, gpuResult.w * gpuResult.h * sizeof(unsigned char));
    // if (err != hipSuccess) {
    //     fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
    //     if (gpuResult.img) hipFree(gpuResult.img);
    //     return(gpuResult);
    // }
    hipError_t err = hipMalloc((void**)&d_hist, 256 * sizeof(int));  // Allocate memory on the GPU
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
        if (gpuResult.img) hipFree(gpuResult.img);
        return(gpuResult);
    }
    err = hipMalloc((void**)&img_A[0], SegSize * sizeof(unsigned char));  // Allocate memory on the GPU
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
        if (gpuResult.img) hipFree(gpuResult.img);
        return(gpuResult);
    }

    err = hipMalloc((void**)&img_A[1], SegSize * sizeof(unsigned char));  // Allocate memory on the GPU
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
        if (gpuResult.img) hipFree(gpuResult.img);
        return(gpuResult);
    }
    
    hipMemset(d_hist, 0, sizeof(int) * 256);

    for (int i=0; i < 2; i++)  {
        int offset = i * SegSize;
        hipStreamCreate(&stream[i]);
        //hipMemcpyAsync(img_A[i], img_in.img+offset, SegSize*sizeof(unsigned char), hipMemcpyHostToDevice, stream[i]);
        histogramGPU<<<((SegSize)/256), 256, 256*sizeof(int), stream[i] >>>(d_hist, img_in.img+offset, SegSize);

        hipMemcpyAsync(t_hist, d_hist, 256*sizeof(int), hipMemcpyDeviceToHost, stream[i]);
    }
    hipDeviceSynchronize();

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("KapCUDA kernel launch error: %s\n", hipGetErrorString(err));
    }

    hipEventRecord(stopCuda);
    hipEventSynchronize(stopCuda);
    hipEventElapsedTime(&millisecondsTransfers, startCuda, stopCuda);

    int *lut = histogram_equalization_prep(t_hist, 256, gpuResult.w * gpuResult.h);

    hipEventRecord(startCuda1, 0);

    printf("\nGPU1 Execution time: %lf seconds\n", millisecondsTransfers/1000.0);
    int *lut_d;
    err = hipMalloc((void**)&lut_d, 256 * sizeof(int));  // Allocate memory on the GPU
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
        if (gpuResult.img) hipFree(gpuResult.img);
        return(gpuResult);
    }

    for (int i=0; i < 2; i++)  {
        int offset = i * SegSize;
        hipMemcpyAsync(lut_d, lut, 256*sizeof(int), hipMemcpyHostToDevice, stream[i]);
        hipBindTexture(0, texRef, lut_d, 256 * sizeof(int));
        histogram_equalization_GPU<<<((SegSize)/256), 256, 256*sizeof(int), stream[i]>>>(gpuResult.img + offset, img_in.img+offset, lut_d, img_in.w, img_in.h/2);

        //hipMemcpyAsync(result.img + offset, gpuResult.img + offset, ((gpuResult.w * gpuResult.h)/2) * sizeof(unsigned char), hipMemcpyDeviceToHost, stream[i]);
    }

    /*for(int i = 0;i < 256; i++) {
        printf("%d ", t_hist[i]);
    }*/

    //hipFree(d_ImgIn);  
    hipFree(d_hist);
    //hipFree(gpuResult.img);

    hipEventRecord(stopCuda1, 0);
    hipUnbindTexture(texRef); // Unbind texture memory
    hipEventSynchronize(stopCuda1);
    hipEventElapsedTime(&millisecondsTransfers1, startCuda1, stopCuda1);

    millisecondsTransfers += millisecondsTransfers1;

    printf("\nGPU Execution time: %lf seconds\n", millisecondsTransfers/1000.0);

    return gpuResult;
}