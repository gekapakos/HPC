#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"

PGM_IMG contrast_enhancement_g(PGM_IMG img_in)  {
    PGM_IMG result;
    int hist[256];

    result.w = img_in.w;
    result.h = img_in.h;
    result.img = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));

    histogram(hist, img_in.img, img_in.h * img_in.w, 256);
    histogram_equalization(result.img,img_in.img,hist,result.w*result.h, 256);

    return result;
}

PGM_IMG contrast_enhancement_GPU(PGM_IMG img_in)  {
    hipEvent_t startCuda, stopCuda, startCudaNew, stopCudaNew;
    PGM_IMG img_gpu;
    PGM_IMG img_cpu;
    float time = 0, timeNew;
    int hist[256];
    int *d_hist, *d_lut;
    unsigned char * d_ImgIn;

    img_cpu.w = img_in.w;
    img_cpu.h = img_in.h;
    img_cpu.img = (unsigned char *)malloc(img_cpu.w * img_cpu.h * sizeof(unsigned char));
    img_gpu.w = img_in.w;
    img_gpu.h = img_in.h;

    hipEventCreate(&startCuda);
    hipEventCreate(&stopCuda);

    hipEventRecord(startCuda);


    hipError_t err = hipMalloc((void **)&img_gpu.img, img_gpu.w * img_gpu.h * sizeof(unsigned char));
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
        if (img_gpu.img) hipFree(img_gpu.img);
        return(img_gpu);
    }
    err = hipMalloc((void **)&d_ImgIn, img_gpu.w * img_gpu.h * sizeof(unsigned char));
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
        if (img_gpu.img) hipFree(img_gpu.img);
        return(img_gpu);
    }
    err = hipMalloc((void**)&d_hist, 256 * sizeof(int));  // Allocate memory on the GPU
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
        if (img_gpu.img) hipFree(img_gpu.img);
        return(img_gpu);
    }
    
    hipMemset(d_hist, 0, sizeof(int) * 256);

    err = hipMemcpy(d_ImgIn, img_in.img, img_cpu.w * img_cpu.h * sizeof(unsigned char), hipMemcpyHostToDevice);  // Copy data from host to device
    

    histogramGPU<<<((img_cpu.w*img_cpu.h)/1024), 1024 >>>(d_hist, d_ImgIn, img_cpu.w, img_cpu.h);

    err = hipMemcpy(hist, d_hist, 256 * sizeof(int), hipMemcpyDeviceToHost);  // Copy data from host to device

    hipEventRecord(stopCuda);
    hipEventSynchronize(stopCuda);
    hipEventElapsedTime(&time, startCuda, stopCuda);

    printf("\nGPU1 Execution time: %lf seconds\n", time/1000.0);
    hipEventCreate(&startCudaNew);
    hipEventCreate(&stopCudaNew);

    int *lut = lut_computation(hist, 256, img_cpu.w * img_cpu.h);

    hipEventRecord(startCudaNew, 0);

    hipMalloc((void **)&d_lut, sizeof(int)*256);

    hipMemcpy(d_lut, lut, sizeof(int)*256, hipMemcpyHostToDevice);  // Copy data from host to device

    histogram_equalization_GPU<<<((img_cpu.w * img_cpu.h)/1024), 1024>>>(img_gpu.img, d_ImgIn, d_lut, img_cpu.w, img_cpu.h);
    hipDeviceSynchronize(); 
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
    }

    err = hipMemcpy(img_cpu.img, img_gpu.img, img_cpu.w * img_cpu.h * sizeof(unsigned char), hipMemcpyDeviceToHost);  // Copy data from host to device

    hipFree(d_ImgIn);  
    hipFree(d_hist);
    hipFree(img_gpu.img);
    hipFree(d_lut);

    hipEventRecord(stopCudaNew, 0);
    hipEventSynchronize(stopCudaNew);
    hipEventElapsedTime(&timeNew, startCudaNew, stopCudaNew);

    time += timeNew;

    printf("\nGPU Execution time: %lf seconds\n", time/1000.0);

    return img_cpu;
}