#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"

PGM_IMG contrast_enhancement_g(PGM_IMG img_in)  {
    PGM_IMG result;
    int hist[256];

    result.w = img_in.w;
    result.h = img_in.h;
    result.img = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));

    histogram(hist, img_in.img, img_in.h * img_in.w, 256);
    histogram_equalization(result.img,img_in.img,hist,result.w*result.h, 256);

    return result;
}

PGM_IMG contrast_enhancement_GPU(PGM_IMG img_in)  {
    hipEvent_t startCuda, stopCuda;
    PGM_IMG gpuResult;
    PGM_IMG result;
    float millisecondsTransfers = 0, time;
    //int hist[256];
    int t_hist[256];
    int *d_hist;
    unsigned char * d_ImgIn;

    result.w = img_in.w;
    result.h = img_in.h;
    //result.img = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));
    
    hipMallocManaged(&gpuResult.img, result.w * result.h * sizeof(unsigned char));

    gpuResult.w = img_in.w;
    gpuResult.h = img_in.h;

    hipEventCreate(&startCuda);
    hipEventCreate(&stopCuda);

    hipEventRecord(startCuda);


    // hipError_t err = hipMalloc((void **)&gpuResult.img, gpuResult.w * gpuResult.h * sizeof(unsigned char));
    // if (err != hipSuccess) {
    //     fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
    //     if (gpuResult.img) hipFree(gpuResult.img);
    //     return(gpuResult);
    // }
    // hipError_t err = hipMalloc((void **)&d_ImgIn, gpuResult.w * gpuResult.h * sizeof(unsigned char));
    // if (err != hipSuccess) {
    //     fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
    //     if (gpuResult.img) hipFree(gpuResult.img);
    //     return(gpuResult);
    // }
    hipError_t err = hipMalloc((void**)&d_hist, 256 * sizeof(int));  // Allocate memory on the GPU
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
        if (gpuResult.img) hipFree(gpuResult.img);
        return(gpuResult);
    }
    
    hipMemset(d_hist, 0, sizeof(int) * 256);

    //err = hipMemcpy(d_ImgIn, img_in.img, gpuResult.w * gpuResult.h * sizeof(unsigned char), hipMemcpyHostToDevice);  // Copy data from host to device
    
    histogramGPU<<<((gpuResult.h*gpuResult.w)/256)+1, 256, 256*sizeof(int) >>>(d_hist, img_in.img, gpuResult.w, gpuResult.h);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDAKap kernel launch error: %s####################\n", hipGetErrorString(err));
    }

    err = hipMemcpy(t_hist, d_hist, 256 * sizeof(int), hipMemcpyDeviceToHost);  // Copy data from host to device

    hipEventRecord(stopCuda);
    hipEventSynchronize(stopCuda);
    hipEventElapsedTime(&millisecondsTransfers, startCuda, stopCuda);

    printf("\nGPU1 Execution time: %lf seconds\n", millisecondsTransfers/1000.0);
    time = histogram_equalization_prep(gpuResult.img, img_in.img, t_hist, gpuResult.w, gpuResult.h, 256, img_in.img);

    time += millisecondsTransfers;

    hipEventRecord(startCuda, 0);

    //err = hipMemcpy(result.img, gpuResult.img, gpuResult.w * gpuResult.h * sizeof(unsigned char), hipMemcpyDeviceToHost);  // Copy data from host to device

    //hipFree(d_ImgIn);  
    hipFree(d_hist);
    //hipFree(gpuResult.img);

    hipEventRecord(stopCuda, 0);
    hipEventSynchronize(stopCuda);
    hipEventElapsedTime(&millisecondsTransfers, startCuda, stopCuda);

    time += millisecondsTransfers;

    printf("\nGPU Execution time: %lf seconds\n", time/1000.0);

    return gpuResult;
}