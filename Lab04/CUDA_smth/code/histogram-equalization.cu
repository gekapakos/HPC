#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"


void histogram(int * hist_out, unsigned char * img_in, int img_size, int nbr_bin) {
    int i;

    // Initialization in cpu
    for (i = 0; i < nbr_bin; i++) {
        hist_out[i] = 0;
    }

    // Constructs the Histogram Vector in gpu
    for (i = 0; i < img_size; i++) {
        //printf("CPUindex: %d, img: %d\n", i, img_in[i]);
        hist_out[img_in[i]]++;
    }
    
}

void histogram_equalization(unsigned char * img_out, unsigned char * img_in, int * hist_in, int img_size, int nbr_bin) {
    int *lut = (int *)malloc(sizeof(int)*nbr_bin);
    int i, cdf, min, d, index;

    /* Construct the LUT by calculating the CDF */
    cdf = 0;
    min = 0;
    i = 0;
    
    // Finds the first value on the Histogram that isn't 0
    while(min == 0) {
        min = hist_in[i++];
    }
    index = i-1;

    // Calculate the look up table (lut)
    for (i = 0; i < index + 1; i++)  {
        lut[i] = 0;
    }

    d = img_size - min;
    for(i = index; i < nbr_bin; i++) {
        cdf += hist_in[i];
        lut[i] = (int)(((float)cdf - min)*255/d + 0.5);
    }

    for(i = 0; i < nbr_bin; i++)  {
       if(lut[i] > 255) {
            lut[i] = 255;
        }
    }
    
    /* Get the result image this is the only part of the function to be run in GPU */
    for(i = 0; i < img_size; i++) {
        img_out[i] = (unsigned char)lut[img_in[i]];
    }
    free(lut);
}

__global__ void histogramGPU(int * hist_out, unsigned char * img_in, int imageW, int imageH) {
    extern __shared__ int sharedMemory[];
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    int tx = threadIdx.x;

    if (tx < 256) {
        sharedMemory[tx] = 0;
    }

    __syncthreads();

    // Constructs the Histogram Vector
    if (index < imageH*imageW)  {
        atomicAdd(&sharedMemory[img_in[index]], 1);
        __syncthreads();
        atomicAdd(&hist_out[tx], sharedMemory[tx]);
    }
    __syncthreads();
}

texture<int, hipTextureType1D, hipReadModeElementType> texRef; // Bind the 1D texture

__global__ void histogram_equalization_GPU(unsigned char * img_out, unsigned char * img_in, int * lut, int imageW, int imageH) {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    int y = index / imageW; // row
    int x = index % imageW; // col
    
    /* Get the result image */

    if ((y * imageW + x) < imageW * imageH)  {
        img_out[index] = tex1Dfetch(texRef, img_in[index]);
    }
    //printf("ABLACK: %d\n", img_out[y*imageW+x]);

}

// __global__ void histogram_equalization_GPU(unsigned char * img_out, unsigned char * img_in, int * lut, int imageW, int imageH) {
//     int index = blockIdx.x*blockDim.x + threadIdx.x;
//     int y = index / imageW; // row
//     int x = index % imageW; // col
//     extern __shared__ unsigned char cuChulain[];
//     /* Get the result image */

//     if (threadIdx.x < 256)  {
//         cuChulain[threadIdx.x] =  lut[threadIdx.x];
//     }
    
//     __syncthreads();

//     if ((y * imageW + x) < imageW * imageH)  {
//         img_out[y*imageW+x] = cuChulain[img_in[y*imageW+x]];
//     }
//     __syncthreads();
//     //printf("ABLACK: %d\n", img_out[y*imageW+x]);

// }

int histogram_equalization_prep(unsigned char * img_out, unsigned char * img_in, int * hist_in, int imageW, int imageH, int nbr_bin, unsigned char * d_ImgIn) {
    int *lut = (int *)malloc(sizeof(int)*nbr_bin);
    int i, cdf, min, d, index, *d_lut;
    int img_size = imageW * imageH;
    float millisecondsTransfers = 0;
    hipEvent_t startCuda, stopCuda;

    /* Construct the LUT by calculating the CDF */
    cdf = 0;
    min = 0;
    i = 0;
    
    // Finds the first value on the Histogram that isn't 0
    while(min == 0) {
        min = hist_in[i++];
    }
    index = i-1;

    // Calculate the look up table (lut)
    for (i = 0; i < index + 1; i++)  {
        lut[i] = 0;
    }

    d = img_size - min;
    for(i = index; i < nbr_bin; i++) {
        cdf += hist_in[i];
        lut[i] = (int)(((float)cdf - min)*255/d + 0.5);
    }
    for(i = 0; i < nbr_bin; i++)  {
        if(lut[i] > 255) {
            // printf("BLACK: %d\n", i);
            lut[i] = 255;
        }
    }

    // hipMalloc((void **)&d_ImgIn, img_size * sizeof(unsigned char));

    // hipMemcpy(d_ImgIn, img_in, img_size * sizeof(unsigned char), hipMemcpyHostToDevice);  // Copy data from host to device

    hipEventCreate(&startCuda);
    hipEventCreate(&stopCuda);

    hipEventRecord(startCuda, 0);

    hipMalloc((void **)&d_lut, sizeof(int)*nbr_bin);

    hipMemcpy(d_lut, lut, sizeof(int)*nbr_bin, hipMemcpyHostToDevice);  // Copy data from host to device

    hipBindTexture(0, texRef, d_lut, 256 * sizeof(int));

    // histogram_equalization_GPU<<<(img_size/256)+1, 256, 256 * sizeof(unsigned char)>>>(img_out, d_ImgIn, d_lut, imageW, imageH);
    histogram_equalization_GPU<<<(img_size/256)+1, 256>>>(img_out, d_ImgIn, d_lut, imageW, imageH);
    hipDeviceSynchronize(); 
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
    }
    hipFree(d_lut);

    hipEventRecord(stopCuda, 0);
    hipEventSynchronize(stopCuda);
    hipUnbindTexture(texRef); // Unbind texture memory
    hipEventElapsedTime(&millisecondsTransfers, startCuda, stopCuda);

    free(lut);
    return(millisecondsTransfers);
}