#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"
/*#define CHECK_CUDA_ERROR(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            cleanup();
        } \
    } while (0)*/

/*void cleanup (void * gpuResult, )  {
    hipFree(gpuResult);
}*/

PGM_IMG contrast_enhancement_g(PGM_IMG img_in)  {
    PGM_IMG result;
    int hist[256];

    result.w = img_in.w;
    result.h = img_in.h;
    result.img = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));

    histogram(hist, img_in.img, img_in.h * img_in.w, 256);
    histogram_equalization(result.img,img_in.img,hist,result.w*result.h, 256);

    // printf("CPU\n");
    // for (int i = 0; i < 256; i++)  {
    //     printf(" %d",hist[i]);
    // }
    return result;
}

PGM_IMG contrast_enhancement_GPU(PGM_IMG img_in)  {
    hipEvent_t startCuda, stopCuda;
    PGM_IMG gpuResult;
    PGM_IMG result;
    float millisecondsTransfers = 0, time;
    //int hist[256];
    int t_hist[256];
    int *d_hist;
    unsigned char * d_ImgIn;

    result.w = img_in.w;
    result.h = img_in.h;
    result.img = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));

    gpuResult.w = img_in.w;
    gpuResult.h = img_in.h;

    hipEventCreate(&startCuda);
    hipEventCreate(&stopCuda);

    hipEventRecord(startCuda);


    hipError_t err = hipMalloc((void **)&gpuResult.img, gpuResult.w * gpuResult.h * sizeof(unsigned char));
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
        if (gpuResult.img) hipFree(gpuResult.img);
        return(gpuResult);
        //cleanup(gpuResult.img, result.img);
    }
    err = hipMalloc((void **)&d_ImgIn, gpuResult.w * gpuResult.h * sizeof(unsigned char));
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
        if (gpuResult.img) hipFree(gpuResult.img);
        return(gpuResult);
        //cleanup(gpuResult.img, result.img);
    }
    err = hipMalloc((void**)&d_hist, 256 * sizeof(int));  // Allocate memory on the GPU
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
        if (gpuResult.img) hipFree(gpuResult.img);
        return(gpuResult);
        //cleanup(gpuResult.img, result.img);
    }
    
    //histogramGPU<<<1, 256>>>(d_hist);
    hipMemset(d_hist, 0, sizeof(int) * 256);

    err = hipMemcpy(d_ImgIn, img_in.img, gpuResult.w * gpuResult.h * sizeof(unsigned char), hipMemcpyHostToDevice);  // Copy data from host to device
    

    histogramGPU<<<((gpuResult.h*gpuResult.w)/256)+1, 256, 256*sizeof(int)>>>(d_hist, d_ImgIn, gpuResult.w, gpuResult.h);

    err = hipMemcpy(t_hist, d_hist, 256 * sizeof(int), hipMemcpyDeviceToHost);  // Copy data from host to device

    /*for (int i = 0; i < 256; i++)  {
        printf("Kapak: %d\n", t_hist[i]);
    }*/

    hipEventRecord(stopCuda);
    hipEventSynchronize(stopCuda);
    hipEventElapsedTime(&millisecondsTransfers, startCuda, stopCuda);

    printf("\nGPU1 Execution time: %lf seconds\n", millisecondsTransfers/1000.0);
    time = histogram_equalization_prep(gpuResult.img, img_in.img, t_hist, gpuResult.w, gpuResult.h, 256, d_ImgIn);

    time += millisecondsTransfers;

    hipEventRecord(startCuda, 0);

    err = hipMemcpy(result.img, gpuResult.img, gpuResult.w * gpuResult.h * sizeof(unsigned char), hipMemcpyDeviceToHost);  // Copy data from host to device

    hipFree(d_ImgIn);  
    hipFree(d_hist);
    hipFree(gpuResult.img);

    hipEventRecord(stopCuda, 0);
    hipEventSynchronize(stopCuda);
    hipEventElapsedTime(&millisecondsTransfers, startCuda, stopCuda);

    time += millisecondsTransfers;

    printf("\nGPU Execution time: %lf seconds\n", time/1000.0);

    return result;
}