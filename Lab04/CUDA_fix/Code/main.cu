#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"

bool run_cpu_gray_test(PGM_IMG img_in, char *out_filename);
bool run_GPU_gray_test(PGM_IMG img_in, char *out_filename);

int main(int argc, char *argv[]){
    PGM_IMG img_ibuf_g;
    struct timespec  tv1, tv2;
    float elapsed_time_CPU;
    bool result;

	if (argc != 3) {
		printf("Run with input file name and output file name as arguments\n");
		exit(1);
	}
	
    printf("Running contrast enhancement for gray-scale images.\n");
    img_ibuf_g = read_pgm(argv[1]);

    result = run_GPU_gray_test(img_ibuf_g, argv[2]);

    free_pgm(img_ibuf_g);

	return 0;
}

bool run_cpu_gray_test(PGM_IMG img_in, char *out_filename)
{
    PGM_IMG img_obuf;

    printf("Starting CPU processing...\n");
    img_obuf = contrast_enhancement_g(img_in);
    if (img_obuf.img == NULL)  {
        free_pgm(img_obuf);
        return(false);
    }
    write_pgm(img_obuf, out_filename);
    free_pgm(img_obuf);
    return(true);
}

bool run_GPU_gray_test(PGM_IMG img_in, char *out_filename)
{
    PGM_IMG img_obuf;
    
    
    printf("Starting CPU processing...\n");
    img_obuf = contrast_enhancement_GPU(img_in);
    if (img_obuf.img == NULL)  {
        free_pgm(img_obuf);
        return(false);
    }
    write_pgm(img_obuf, out_filename);
    free_gpu_pgm(img_obuf);
    return(true);
}

bool run_gpu_gray_test(PGM_IMG img_in, char *out_filename)  {
    PGM_IMG img_obuf;
    
    
    printf("Starting CPU processing...\n");
    img_obuf = contrast_enhancement_GPU(img_in);
    if (img_obuf.img == NULL)  {
        return(false);
    }
    hipDeviceSynchronize(); 
    write_pgm(img_obuf, out_filename);
    free_gpu_pgm(img_obuf);
    return(true);
}

PGM_IMG read_pgm(const char * path){
    FILE * in_file;
    char sbuf[256];
    
    
    PGM_IMG result;
    int v_max;//, i;
    in_file = fopen(path, "r");
    if (in_file == NULL){
        printf("Input file not found!\n");
        exit(1);
    }
    
    fscanf(in_file, "%s", sbuf); /*Skip the magic number*/
    fscanf(in_file, "%d",&result.w);
    fscanf(in_file, "%d",&result.h);
    fscanf(in_file, "%d\n",&v_max);
    printf("Image size: %d x %d\n", result.w, result.h);
    
    hipMallocManaged(&result.img, result.w * result.h * sizeof(unsigned char));
        
    fread(result.img,sizeof(unsigned char), result.w*result.h, in_file);    
    fclose(in_file);
    
    return result;
}

void write_pgm(PGM_IMG img, const char * path){
    FILE * out_file;
    out_file = fopen(path, "wb");
    fprintf(out_file, "P5\n");
    fprintf(out_file, "%d %d\n255\n",img.w, img.h);
    fwrite(img.img,sizeof(unsigned char), img.w*img.h, out_file);
    fclose(out_file);
}

void free_pgm(PGM_IMG img)
{
    hipFree(img.img);
}

void free_gpu_pgm(PGM_IMG img)
{
    hipFree(img.img);
}