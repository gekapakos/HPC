#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"

PGM_IMG contrast_enhancement_g(PGM_IMG img_in)  {
    PGM_IMG result;
    int hist[256];

    result.w = img_in.w;
    result.h = img_in.h;
    result.img = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));

    histogram(hist, img_in.img, img_in.h * img_in.w, 256);
    histogram_equalization(result.img,img_in.img,hist,result.w*result.h, 256);

    return result;
}

PGM_IMG contrast_enhancement_GPU(PGM_IMG img_in)  {
    hipEvent_t startCuda, stopCuda;
    hipError_t err;
    PGM_IMG gpuResult;
    PGM_IMG result;
    float millisecondsTransfers = 0, time;
    //int hist[256];
    int t_hist[256];
    int *d_hist;
    unsigned char * d_ImgIn;

    result.w = img_in.w;
    result.h = img_in.h;
    // result.img = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));
    err = hipHostAlloc((void**)&result.img, result.w * result.h * sizeof(unsigned char), hipHostMallocDefault);
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Host Alloc error: %s\n", hipGetErrorString(err));
        if (result.img) hipHostFree(result.img);
        return(result);
    }

    gpuResult.w = img_in.w;
    gpuResult.h = img_in.h;

    hipEventCreate(&startCuda);
    hipEventCreate(&stopCuda);

    hipEventRecord(startCuda);


    err = hipMalloc((void **)&gpuResult.img, gpuResult.w * gpuResult.h * sizeof(unsigned char));
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
        if (gpuResult.img) hipFree(gpuResult.img);
        return(gpuResult);
    }
    err = hipMalloc((void **)&d_ImgIn, gpuResult.w * gpuResult.h * sizeof(unsigned char));
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
        if (gpuResult.img) hipFree(gpuResult.img);
        return(gpuResult);
    }
    err = hipMalloc((void**)&d_hist, 256 * sizeof(int));  // Allocate memory on the GPU
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
        if (gpuResult.img) hipFree(gpuResult.img);
        return(gpuResult);
    }
    
    hipMemset(d_hist, 0, sizeof(int) * 256);

    err = hipMemcpy(d_ImgIn, img_in.img, gpuResult.w * gpuResult.h * sizeof(unsigned char), hipMemcpyHostToDevice);  // Copy data from host to device
    

    histogramGPU<<<((gpuResult.h*gpuResult.w)/256)+1, 256, 256*sizeof(int) >>>(d_hist, d_ImgIn, gpuResult.w, gpuResult.h);

    err = hipMemcpy(t_hist, d_hist, 256 * sizeof(int), hipMemcpyDeviceToHost);  // Copy data from host to device

    hipEventRecord(stopCuda);
    hipEventSynchronize(stopCuda);
    hipEventElapsedTime(&millisecondsTransfers, startCuda, stopCuda);

    printf("\nGPU1 Execution time: %lf seconds\n", millisecondsTransfers/1000.0);
    time = histogram_equalization_prep(gpuResult.img, img_in.img, t_hist, gpuResult.w, gpuResult.h, 256, d_ImgIn);

    time += millisecondsTransfers;

    hipEventRecord(startCuda, 0);

    err = hipMemcpy(result.img, gpuResult.img, gpuResult.w * gpuResult.h * sizeof(unsigned char), hipMemcpyDeviceToHost);  // Copy data from host to device

    hipFree(d_ImgIn);  
    hipFree(d_hist);
    hipFree(gpuResult.img);

    hipEventRecord(stopCuda, 0);
    hipEventSynchronize(stopCuda);
    hipEventElapsedTime(&millisecondsTransfers, startCuda, stopCuda);

    time += millisecondsTransfers;

    printf("\nGPU Execution time: %lf seconds\n", time/1000.0);
    printf("\nCPU Execution time: 0.0000");

    return result;
}